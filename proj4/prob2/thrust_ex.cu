#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

using namespace thrust::placeholders;

int N = 1000000000;
double step = 1.0/(double)N;

struct function {
    __host__ __device__
    double operator()(const double& x) const {
        return 4 / (1 + x * x);
    }
};

int main ()
{
    clock_t tStart = clock();
    thrust::device_vector<double> Y(N);
    thrust::sequence(Y.begin(), Y.end(), step*0.5, step);
    thrust::transform(Y.begin(), Y.end(), Y.begin(), function());
    thrust::transform(Y.begin(), Y.end(), Y.begin(), step * _1);
    double integral = thrust::reduce(Y.begin(), Y.end());
    clock_t tEnd = clock();
    printf("Execution Time : %fms\n", (double)(tEnd-tStart)/CLOCKS_PER_SEC*1000);
    printf("pi=%.10lf\n", integral);
    return 0;
}
